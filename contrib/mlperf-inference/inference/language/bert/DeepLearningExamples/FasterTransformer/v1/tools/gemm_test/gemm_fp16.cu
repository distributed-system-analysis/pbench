#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cstdio>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <ctime>
#include <unistd.h>
#include <sys/time.h>
#include "common.h"
using namespace std;
double diffTime(timeval start, timeval end)
{
  return (end.tv_sec - start.tv_sec) * 1000 + (end.tv_usec - start.tv_usec) * 0.001;
}
int main(int argc, char* argv[])
{
  FILE* fd = fopen("gemm_config.in", "w");
  if(fd == NULL)
  {
    printf("Cannot write to file gemm_config.in\n");
    return 0;
  }
  struct hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("Device %s\n", prop.name);

  const int batch_size = atoi(argv[1]);
  const int seq_len = atoi(argv[2]);
  const int head_num = atoi(argv[3]);
  const int size_per_head = atoi(argv[4]);

  const int gemm_num = 5;
  int M[gemm_num];
  int N[gemm_num];
  int K[gemm_num];
  int batchCount[gemm_num] = {1,1,1,1,1};
  char mess[gemm_num][256];
  
  //gemm1 
  M[0] = batch_size * seq_len;
  K[0] = head_num * size_per_head;
  N[0] = K[0];
  strcpy(mess[0], "from_tensor * weightQ/K/V, attr * output_kernel");

  //gemm2
  M[1] = M[0];
  K[1] = K[0];
  N[1] = 4 * N[0];
  strcpy(mess[1], "attr_output * inter_kernel");

  //gemm3
  M[2] = M[0];
  K[2] = 4 * K[0];
  N[2] = N[0];
  strcpy(mess[2], "inter_matmul * output_kernel");

  M[3] = seq_len;
  N[3] = seq_len;
  K[3] = size_per_head;
  batchCount[3] = batch_size * head_num;
  strcpy(mess[3], "attention batched Gemm1");

  M[4] = seq_len;
  N[4] = size_per_head; 
  K[4] = seq_len;
  batchCount[4] = batch_size * head_num;
  strcpy(mess[4], "attention batched Gemm2");

  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);

  typedef __half T;
  hipDataType AType = HIP_R_16F;
  hipDataType BType = HIP_R_16F;
  hipDataType CType = HIP_R_16F;
  hipDataType computeType = HIP_R_16F;
  const int ites = 100;
  struct timeval start, end;
  int startAlgo = (int)CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  int endAlgo = (int)CUBLAS_GEMM_ALGO15_TENSOR_OP;
  T alpha = (T)1.0f;
  T beta = (T)0.0f;

  printf("***FP16 Gemm Testing***\n");
  for(int i = 0; i < gemm_num; ++i)
  {
    int m = M[i], n = N[i], k = K[i];
    printf("\n-----------------------------\n");
    printf("GEMM test %d: [M: %d, K: %d, N: %d] %s\n", i, m, k, n, mess[i]);
    T* d_A;
    T* d_B;
    T* d_C;
    check_cuda_error(hipMalloc((void**)&d_A, sizeof(T) * m * k * batchCount[i]));
    check_cuda_error(hipMalloc((void**)&d_B, sizeof(T) * k * n * batchCount[i]));
    check_cuda_error(hipMalloc((void**)&d_C, sizeof(T) * m * n * batchCount[i]));

    float exec_time = 99999.0f;
    int fast_algo = 0;
    for(int algo = startAlgo; algo <= endAlgo; algo++)
    {
      hipDeviceSynchronize();
      gettimeofday(&start, NULL);
      for(int ite = 0; ite < ites; ++ite)
      {
        if(i < 3)
        {
          check_cuda_error(hipblasGemmEx(cublas_handle, 
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, m, k, 
                &alpha, 
                d_B, BType, n, 
                d_A, AType, k, 
                &beta, 
                d_C, CType, n, 
                computeType, 
                static_cast<hipblasGemmAlgo_t>(algo)));
        }
        else if(i == 3)
        {
          check_cuda_error(hipblasGemmStridedBatchedEx(cublas_handle,
                HIPBLAS_OP_T, HIPBLAS_OP_N,
                seq_len, seq_len, size_per_head,
                &alpha,
                d_B, BType, size_per_head, seq_len * size_per_head,
                d_A, AType, size_per_head, seq_len * size_per_head,
                &beta,
                d_C, CType, seq_len, seq_len * seq_len,
                batch_size * head_num,
                computeType,
                static_cast<hipblasGemmAlgo_t>(algo)));
        }
        else
        {
          check_cuda_error(hipblasGemmStridedBatchedEx(cublas_handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                size_per_head, seq_len, seq_len,
                &alpha,
                d_B, BType, size_per_head, seq_len * size_per_head,
                d_A, AType, seq_len, seq_len * seq_len,
                &beta,
                d_C, CType, size_per_head, seq_len * size_per_head,
                batch_size * head_num,
                computeType,
                static_cast<hipblasGemmAlgo_t>(algo)));
        }
      }
      hipDeviceSynchronize();
      gettimeofday(&end, NULL);
      printf("algo_%d costs %.3fms \n", algo, diffTime(start, end) / ites);
      if(diffTime(start, end) / ites < exec_time)
      {
        exec_time = diffTime(start, end) / ites;
        fast_algo = algo;
      }
    }
    printf("fast_algo %d costs %.3f ms\n", fast_algo, exec_time);
    fprintf(fd, "%d\n", fast_algo);
  }

}

