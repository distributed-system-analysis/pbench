#include "hip/hip_runtime.h"
// Copyright (c) 2018-2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <string>

#include "src/core/model_config.h"
#include "src/core/model_config.pb.h"
#include "src/custom/sdk/custom_instance.h"

#ifdef TRTIS_ENABLE_GPU
#include <hip/hip_runtime.h>
#include "src/core/model_config_cuda.h"
#include "src/custom/addsub/kernel.h"
#endif  // TRTIS_ENABLE_GPU

#define LOG_ERROR std::cerr
#define LOG_INFO std::cout

// This custom backend takes two input tensors (any shape but must
// have the same shape) and produces two output tensors (with same
// shape as the inputs). All tensors must be the same data-type,
// either INT32 or FP32. The input tensors must be named "INPUT0" and
// "INPUT1". The output tensors must be named "OUTPUT0" and
// "OUTPUT1". This backend does element-wise operation to produce:
//
//   OUTPUT0 = INPUT0 + INPUT1
//   OUTPUT1 = INPUT0 - INPUT1
//

namespace nvidia { namespace inferenceserver { namespace custom {
namespace addsub {

// Context object. All state must be kept in this object.
class Context : public CustomInstance {
 public:
  Context(
      const std::string& instance_name, const ModelConfig& config,
      const int gpu_device);
  ~Context();

  // Initialize the context. Validate that the model configuration,
  // etc. is something that we can handle.
  int Init();

  // Perform custom execution on the payloads.
  int Execute(
      const uint32_t payload_cnt, CustomPayload* payloads,
      CustomGetNextInputFn_t input_fn, CustomGetOutputFn_t output_fn);

 private:
#ifdef TRTIS_ENABLE_GPU
  int FreeCudaBuffers();
  int AllocateCudaBuffers(size_t byte_size);

  int GetInputTensorGPU(
      CustomGetNextInputFn_t input_fn, void* input_context, const char* name,
      const size_t expected_byte_size, uint8_t* input);
  int ExecuteGPU(
      const uint32_t payload_cnt, CustomPayload* payloads,
      CustomGetNextInputFn_t input_fn, CustomGetOutputFn_t output_fn);
#endif  // TRTIS_ENABLE_GPU

  int GetInputTensorCPU(
      CustomGetNextInputFn_t input_fn, void* input_context, const char* name,
      const size_t expected_byte_size, std::vector<uint8_t>* input);
  int ExecuteCPU(
      const uint32_t payload_cnt, CustomPayload* payloads,
      CustomGetNextInputFn_t input_fn, CustomGetOutputFn_t output_fn);

  // The data-type of the input and output tensors. Must be either
  // INT32 or FP32.
  DataType datatype_ = DataType::TYPE_INVALID;

#ifdef TRTIS_ENABLE_GPU
  // CUDA memory buffers for input and output tensors.
  size_t cuda_buffer_byte_size_;
  uint8_t* cuda_input0_;
  uint8_t* cuda_input1_;
  uint8_t* cuda_output_;

  // The contexts executing on a GPU, the CUDA stream to use for the
  // execution.
  hipStream_t stream_;
#endif  // TRTIS_ENABLE_GPU

  // Local error codes
  const int kGpuNotSupported = RegisterError("execution on GPU not supported");
  const int kInputOutputShape = RegisterError(
      "model must have two inputs and two outputs with the same shape");
  const int kInputName =
      RegisterError("model inputs must be named 'INPUT0' and 'INPUT1'");
  const int kOutputName =
      RegisterError("model outputs must be named 'OUTPUT0' and 'OUTPUT1'");
  const int kInputOutputDataType = RegisterError(
      "model inputs and outputs must have TYPE_INT32 or TYPE_FP32 data-type");
  const int kInputContents = RegisterError("unable to get input tensor values");
  const int kInputSize = RegisterError("unexpected size for input tensor");
  const int kOutputBuffer =
      RegisterError("unable to get buffer for output tensor values");
  const int kCudaDevice = RegisterError("hipSetDevice failed");
  const int kCudaMalloc = RegisterError("hipMalloc failed");
  const int kCudaMemcpy = RegisterError("hipMemcpy failed");
  const int kCudaExecute = RegisterError("cuda execution failed");
  const int kCudaStream = RegisterError("failed to create CUDA stream");
};

Context::Context(
    const std::string& instance_name, const ModelConfig& model_config,
    const int gpu_device)
    : CustomInstance(instance_name, model_config, gpu_device)
#ifdef TRTIS_ENABLE_GPU
      ,
      cuda_buffer_byte_size_(0), cuda_input0_(nullptr), cuda_input1_(nullptr),
      cuda_output_(nullptr), stream_(nullptr)
#endif  // TRTIS_ENABLE_GPU
{
}

Context::~Context()
{
#ifdef TRTIS_ENABLE_GPU
  FreeCudaBuffers();

  if (stream_ != nullptr) {
    hipError_t cuerr = hipStreamDestroy(stream_);
    if (cuerr != hipSuccess) {
      LOG_ERROR << "Failed to destroy cuda stream: "
                << hipGetErrorString(cuerr);
    }
    stream_ = nullptr;
  }
#endif  // TRTIS_ENABLE_GPU
}

#ifdef TRTIS_ENABLE_GPU
int
Context::FreeCudaBuffers()
{
  if (cuda_input0_ != nullptr) {
    hipError_t cuerr = hipFree(cuda_input0_);
    if (cuerr != hipSuccess) {
      LOG_ERROR << "Failed to free cuda memory: " << hipGetErrorString(cuerr);
    }
    cuda_input0_ = nullptr;
  }
  if (cuda_input1_ != nullptr) {
    hipError_t cuerr = hipFree(cuda_input1_);
    if (cuerr != hipSuccess) {
      LOG_ERROR << "Failed to free cuda memory: " << hipGetErrorString(cuerr);
    }
    cuda_input1_ = nullptr;
  }
  if (cuda_output_ != nullptr) {
    hipError_t cuerr = hipFree(cuda_output_);
    if (cuerr != hipSuccess) {
      LOG_ERROR << "Failed to free cuda memory: " << hipGetErrorString(cuerr);
    }
    cuda_output_ = nullptr;
  }

  cuda_buffer_byte_size_ = 0;
  return ErrorCodes::Success;
}

int
Context::AllocateCudaBuffers(size_t byte_size)
{
  hipError_t cuerr;

  FreeCudaBuffers();

  // Allocate GPU memory buffers large enough for each input and
  // output. For performance we allocate once during initialization
  // instead of doing it each time we execute.
  cuerr = hipMalloc(&cuda_input0_, byte_size);
  if (cuerr != hipSuccess) {
    LOG_ERROR << "unable to allocate memory for addsub: "
              << hipGetErrorString(cuerr);
    return kCudaMalloc;
  }
  cuerr = hipMalloc(&cuda_input1_, byte_size);
  if (cuerr != hipSuccess) {
    LOG_ERROR << "unable to allocate memory for addsub: "
              << hipGetErrorString(cuerr);
    return kCudaMalloc;
  }
  cuerr = hipMalloc(&cuda_output_, byte_size);
  if (cuerr != hipSuccess) {
    LOG_ERROR << "unable to allocate memory for addsub: "
              << hipGetErrorString(cuerr);
    return kCudaMalloc;
  }

  cuda_buffer_byte_size_ = byte_size;
  return ErrorCodes::Success;
}
#endif  // TRTIS_ENABLE_GPU

int
Context::Init()
{
  // There must be two inputs that have the same shape. The shape can
  // be anything (including having wildcard, -1, dimensions) since we
  // are just going to do an element-wise add and an element-wise
  // subtract. The input data-type must be INT32 or FP32. The inputs
  // must be named INPUT0 and INPUT1.
  if (model_config_.input_size() != 2) {
    return kInputOutputShape;
  }
  if (!CompareDims(
          model_config_.input(0).dims(), model_config_.input(1).dims())) {
    return kInputOutputShape;
  }

  datatype_ = model_config_.input(0).data_type();
  if (((datatype_ != DataType::TYPE_INT32) &&
       (datatype_ != DataType::TYPE_FP32)) ||
      (model_config_.input(1).data_type() != datatype_)) {
    return kInputOutputDataType;
  }
  if ((model_config_.input(0).name() != "INPUT0") ||
      (model_config_.input(1).name() != "INPUT1")) {
    return kInputName;
  }

  // There must be two outputs that have the same shape as the
  // inputs. The output data-type must be the same as the input
  // data-type. The outputs must be named OUTPUT0 and OUTPUT1.
  if (model_config_.output_size() != 2) {
    return kInputOutputShape;
  }
  if (!CompareDims(
          model_config_.output(0).dims(), model_config_.output(1).dims()) ||
      !CompareDims(
          model_config_.output(0).dims(), model_config_.input(0).dims())) {
    return kInputOutputShape;
  }
  if ((model_config_.output(0).data_type() != datatype_) ||
      (model_config_.output(1).data_type() != datatype_)) {
    return kInputOutputDataType;
  }
  if ((model_config_.output(0).name() != "OUTPUT0") ||
      (model_config_.output(1).name() != "OUTPUT1")) {
    return kOutputName;
  }

  // Additional initialization if executing on the GPU...
  if (gpu_device_ != CUSTOM_NO_GPU_DEVICE) {
#ifndef TRTIS_ENABLE_GPU
    return kGpuNotSupported;
#else
    // Very important to set the CUDA device before performing any
    // CUDA API calls. The device is maintained per-CPU-thread, and
    // the same CPU thread will always be used with this instance of
    // the backend, so only need to set the device once.
    hipError_t cuerr = hipSetDevice(gpu_device_);
    if (cuerr != hipSuccess) {
      LOG_ERROR << "failed to set CUDA device to " << gpu_device_ << ": "
                << hipGetErrorString(cuerr);
      return kCudaDevice;
    }

    // Create a CUDA stream for this context so that it executes
    // independently of other instances of this backend.
    const int cuda_stream_priority =
        GetCudaStreamPriority(model_config_.optimization().priority());
    cuerr = hipStreamCreateWithPriority(
        &stream_, hipStreamDefault, cuda_stream_priority);
    if (cuerr != hipSuccess) {
      LOG_ERROR << "unable to create stream for addsub backend: "
                << hipGetErrorString(cuerr);
      return kCudaStream;
    }
#endif  // !TRTIS_ENABLE_GPU
  }

  return ErrorCodes::Success;
}

namespace {

template <typename T>
void
AddForType(uint64_t cnt, uint8_t* in0, uint8_t* in1, uint8_t* out)
{
  T* output = reinterpret_cast<T*>(out);
  T* input0 = reinterpret_cast<T*>(in0);
  T* input1 = reinterpret_cast<T*>(in1);
  for (uint64_t i = 0; i < cnt; ++i) {
    output[i] = input0[i] + input1[i];
  }
}

template <typename T>
void
SubForType(uint64_t cnt, uint8_t* in0, uint8_t* in1, uint8_t* out)
{
  T* output = reinterpret_cast<T*>(out);
  T* input0 = reinterpret_cast<T*>(in0);
  T* input1 = reinterpret_cast<T*>(in1);
  for (uint64_t i = 0; i < cnt; ++i) {
    output[i] = input0[i] - input1[i];
  }
}

}  // namespace

int
Context::GetInputTensorCPU(
    CustomGetNextInputFn_t input_fn, void* input_context, const char* name,
    const size_t expected_byte_size, std::vector<uint8_t>* input)
{
  // The values for an input tensor are not necessarily in one
  // contiguous chunk, so we copy the chunks into 'input' vector. A
  // more performant solution would attempt to use the input tensors
  // in-place instead of having this copy.
  uint64_t total_content_byte_size = 0;

  while (true) {
    const void* content;
    uint64_t content_byte_size = expected_byte_size;
    if (!input_fn(input_context, name, &content, &content_byte_size)) {
      return kInputContents;
    }

    // If 'content' returns nullptr we have all the input.
    if (content == nullptr) {
      break;
    }

    // If the total amount of content received exceeds what we expect
    // then something is wrong.
    total_content_byte_size += content_byte_size;
    if (total_content_byte_size > expected_byte_size) {
      return kInputSize;
    }

    size_t content_elements = content_byte_size / sizeof(uint8_t);
    input->insert(
        input->end(), static_cast<const uint8_t*>(content),
        static_cast<const uint8_t*>(content) + content_elements);
  }

  // Make sure we end up with exactly the amount of input we expect.
  if (total_content_byte_size != expected_byte_size) {
    return kInputSize;
  }

  return ErrorCodes::Success;
}

int
Context::ExecuteCPU(
    const uint32_t payload_cnt, CustomPayload* payloads,
    CustomGetNextInputFn_t input_fn, CustomGetOutputFn_t output_fn)
{
  // Each payload represents a related set of inputs and required
  // outputs. Each payload may have a different batch size. The total
  // batch-size of all payloads will not exceed the max-batch-size
  // specified in the model configuration.
  if (payload_cnt == 0) {
    return ErrorCodes::Success;
  }

  // For performance, we would typically execute all payloads together
  // as a single batch by first gathering the inputs from across the
  // payloads and then scattering the outputs across the payloads.
  // Here, for simplicity and clarity, we instead process each payload
  // separately.

  // Make sure all inputs have the same shape. We need to do this
  // check for every request to support variable-size input tensors
  // (otherwise the checks of the model configuration in Init() would
  // be sufficient). The scheduler will ensure that all payloads have
  // consistent shape for all inputs so we only need to check that the
  // first payload INPUT0 and INPUT1 are the same shape.

  if (payloads[0].input_cnt != 2) {
    // Should never hit this case since inference server will ensure
    // correct number of inputs...
    return kInputOutputShape;
  }

  std::vector<int64_t> shape(
      payloads[0].input_shape_dims[0],
      payloads[0].input_shape_dims[0] + payloads[0].input_shape_dim_cnts[0]);
  std::vector<int64_t> shape1(
      payloads[0].input_shape_dims[1],
      payloads[0].input_shape_dims[1] + payloads[0].input_shape_dim_cnts[1]);
  if (shape != shape1) {
    return kInputOutputShape;
  }

  const uint64_t batch1_element_count = GetElementCount(shape);
  const uint64_t batch1_byte_size =
      batch1_element_count * GetDataTypeByteSize(datatype_);

  int err;
  for (uint32_t pidx = 0; pidx < payload_cnt; ++pidx) {
    CustomPayload& payload = payloads[pidx];

    // For this payload the expected size of the input and output
    // tensors is determined by the batch-size of this payload.
    const uint64_t batchn_element_count =
        payload.batch_size * batch1_element_count;
    const uint64_t batchn_byte_size = payload.batch_size * batch1_byte_size;

    // Get the input tensors.
    std::vector<uint8_t> input0;
    err = GetInputTensorCPU(
        input_fn, payload.input_context, "INPUT0", batchn_byte_size, &input0);
    if (err != ErrorCodes::Success) {
      payload.error_code = err;
      continue;
    }

    std::vector<uint8_t> input1;
    err = GetInputTensorCPU(
        input_fn, payload.input_context, "INPUT1", batchn_byte_size, &input1);
    if (err != ErrorCodes::Success) {
      payload.error_code = err;
      continue;
    }

    // The output shape is [payload-batch-size, shape] if the model
    // configuration supports batching, or just [shape] if the
    // model configuration does not support batching.
    std::vector<int64_t> output_shape;
    if (model_config_.max_batch_size() != 0) {
      output_shape.push_back(payload.batch_size);
    }
    output_shape.insert(output_shape.end(), shape.begin(), shape.end());

    // For each requested output get the buffer to hold the output
    // values and calculate the sum/difference directly into that
    // buffer.
    for (uint32_t oidx = 0; oidx < payload.output_cnt; ++oidx) {
      const char* output_name = payload.required_output_names[oidx];

      void* obuffer;
      if (!output_fn(
              payload.output_context, output_name, output_shape.size(),
              &output_shape[0], batchn_byte_size, &obuffer)) {
        payload.error_code = kOutputBuffer;
        break;
      }

      // If no error but the 'obuffer' is returned as nullptr, then
      // skip writing this output.
      if (obuffer == nullptr) {
        continue;
      }

      if (!strncmp(output_name, "OUTPUT0", strlen("OUTPUT0"))) {
        if (datatype_ == DataType::TYPE_INT32) {
          AddForType<int32_t>(
              batchn_element_count, &input0[0], &input1[0],
              reinterpret_cast<uint8_t*>(obuffer));
        } else {
          AddForType<float>(
              batchn_element_count, &input0[0], &input1[0],
              reinterpret_cast<uint8_t*>(obuffer));
        }
      } else {
        if (datatype_ == DataType::TYPE_INT32) {
          SubForType<int32_t>(
              batchn_element_count, &input0[0], &input1[0],
              reinterpret_cast<uint8_t*>(obuffer));
        } else {
          SubForType<float>(
              batchn_element_count, &input0[0], &input1[0],
              reinterpret_cast<uint8_t*>(obuffer));
        }
      }
    }
  }

  return ErrorCodes::Success;
}

#ifdef TRTIS_ENABLE_GPU
int
Context::GetInputTensorGPU(
    CustomGetNextInputFn_t input_fn, void* input_context, const char* name,
    const size_t expected_byte_size, uint8_t* input)
{
  // The values for an input tensor are not necessarily in one
  // contiguous chunk, so we copy the chunks into 'input', which
  // points to CUDA memory.
  uint64_t total_content_byte_size = 0;

  while (true) {
    const void* content;
    uint64_t content_byte_size = expected_byte_size;
    if (!input_fn(input_context, name, &content, &content_byte_size)) {
      return kInputContents;
    }

    // If 'content' returns nullptr we have all the input.
    if (content == nullptr) {
      break;
    }

    // If the total amount of content received exceeds what we expect
    // then something is wrong.
    if ((total_content_byte_size + content_byte_size) > expected_byte_size) {
      return kInputSize;
    }

    hipError_t cuerr = hipMemcpyAsync(
        reinterpret_cast<char*>(input) + total_content_byte_size, content,
        content_byte_size, hipMemcpyHostToDevice, stream_);
    if (cuerr != hipSuccess) {
      LOG_ERROR << "failed to copy input values to GPU for addsub: "
                << hipGetErrorString(cuerr);
      return kCudaMemcpy;
    }

    total_content_byte_size += content_byte_size;
  }

  // Make sure we end up with exactly the amount of input we expect.
  if (total_content_byte_size != expected_byte_size) {
    return kInputSize;
  }

  return ErrorCodes::Success;
}

int
Context::ExecuteGPU(
    const uint32_t payload_cnt, CustomPayload* payloads,
    CustomGetNextInputFn_t input_fn, CustomGetOutputFn_t output_fn)
{
  // Each payload represents a related set of inputs and required
  // outputs. Each payload may have a different batch size. The total
  // batch-size of all payloads will not exceed the max-batch-size
  // specified in the model configuration.
  if (payload_cnt == 0) {
    return ErrorCodes::Success;
  }

  // For performance, we would typically execute all payloads together
  // as a single batch by first gathering the inputs from across the
  // payloads and then scattering the outputs across the payloads.
  // Here, for simplicity and clarity, we instead process each payload
  // separately.

  // Make sure all inputs have the same shape. We need to do this
  // check for every request to support variable-size input tensors
  // (otherwise the checks of the model configuration in Init() would
  // be sufficient). The scheduler will ensure that all payloads have
  // consistent shape for all inputs so we only need to check that the
  // first payload INPUT0 and INPUT1 are the same shape.

  if (payloads[0].input_cnt != 2) {
    // Should never hit this case since inference server will ensure
    // correct number of inputs...
    return kInputOutputShape;
  }

  std::vector<int64_t> shape(
      payloads[0].input_shape_dims[0],
      payloads[0].input_shape_dims[0] + payloads[0].input_shape_dim_cnts[0]);
  std::vector<int64_t> shape1(
      payloads[0].input_shape_dims[1],
      payloads[0].input_shape_dims[1] + payloads[0].input_shape_dim_cnts[1]);
  if (shape != shape1) {
    return kInputOutputShape;
  }

  const uint64_t batch1_element_count = GetElementCount(shape);
  const uint64_t batch1_byte_size =
      batch1_element_count * GetDataTypeByteSize(datatype_);

  int err;
  for (uint32_t pidx = 0; pidx < payload_cnt; ++pidx) {
    CustomPayload& payload = payloads[pidx];

    // For this payload the expected size of the input and output
    // tensors is determined by the batch-size of this payload.
    const uint64_t batchn_element_count =
        payload.batch_size * batch1_element_count;
    const uint64_t batchn_byte_size = payload.batch_size * batch1_byte_size;

    // Make sure the CUDA memory buffers are large enough for this
    // payload. If not increase their size.
    if (batchn_byte_size > cuda_buffer_byte_size_) {
      err = AllocateCudaBuffers(batchn_byte_size);
      if (err != ErrorCodes::Success) {
        payload.error_code = err;
        continue;
      }
    }

    // Copy the input tensors into the appropriate CUDA memory buffer.
    err = GetInputTensorGPU(
        input_fn, payload.input_context, "INPUT0", batchn_byte_size,
        cuda_input0_);
    if (err != ErrorCodes::Success) {
      payload.error_code = err;
      continue;
    }

    err = GetInputTensorGPU(
        input_fn, payload.input_context, "INPUT1", batchn_byte_size,
        cuda_input1_);
    if (err != ErrorCodes::Success) {
      payload.error_code = err;
      continue;
    }

    // The output shape is [payload-batch-size, shape] if the model
    // configuration supports batching, or just [shape] if the
    // model configuration does not support batching.
    std::vector<int64_t> output_shape;
    if (model_config_.max_batch_size() != 0) {
      output_shape.push_back(payload.batch_size);
    }
    output_shape.insert(output_shape.end(), shape.begin(), shape.end());

    // For each requested output calculate the sum/difference directly
    // into the CUDA output buffer and then copy out.
    for (uint32_t oidx = 0; oidx < payload.output_cnt; ++oidx) {
      const char* output_name = payload.required_output_names[oidx];

      void* obuffer;
      if (!output_fn(
              payload.output_context, output_name, output_shape.size(),
              &output_shape[0], batchn_byte_size, &obuffer)) {
        payload.error_code = kOutputBuffer;
        break;
      }

      // If no error but the 'obuffer' is returned as nullptr, then
      // skip writing this output.
      if (obuffer == nullptr) {
        continue;
      }

      int block_size = std::min(batchn_element_count, (uint64_t)1024);
      int grid_size = (batchn_element_count + block_size - 1) / block_size;
      if (!strncmp(output_name, "OUTPUT0", strlen("OUTPUT0"))) {
        if (datatype_ == DataType::TYPE_INT32) {
          VecAddInt32<<<grid_size, block_size, 0, stream_>>>(
              reinterpret_cast<int32_t*>(cuda_input0_),
              reinterpret_cast<int32_t*>(cuda_input1_),
              reinterpret_cast<int32_t*>(cuda_output_), batchn_element_count);
        } else {
          VecAddFp32<<<grid_size, block_size, 0, stream_>>>(
              reinterpret_cast<float*>(cuda_input0_),
              reinterpret_cast<float*>(cuda_input1_),
              reinterpret_cast<float*>(cuda_output_), batchn_element_count);
        }
      } else {
        if (datatype_ == DataType::TYPE_INT32) {
          VecSubInt32<<<grid_size, block_size, 0, stream_>>>(
              reinterpret_cast<int32_t*>(cuda_input0_),
              reinterpret_cast<int32_t*>(cuda_input1_),
              reinterpret_cast<int32_t*>(cuda_output_), batchn_element_count);
        } else {
          VecSubFp32<<<grid_size, block_size, 0, stream_>>>(
              reinterpret_cast<float*>(cuda_input0_),
              reinterpret_cast<float*>(cuda_input1_),
              reinterpret_cast<float*>(cuda_output_), batchn_element_count);
        }
      }

      hipError_t cuerr = hipGetLastError();
      if (cuerr != hipSuccess) {
        LOG_ERROR << "failed to launch kernel: " << hipGetErrorString(cuerr)
                  << std::endl;
        payload.error_code = kCudaExecute;
        break;
      }

      cuerr = hipMemcpyAsync(
          obuffer, cuda_output_, batchn_byte_size, hipMemcpyDeviceToHost,
          stream_);
      if (cuerr != hipSuccess) {
        LOG_ERROR << "failed to copy output values from GPU for addsub: "
                  << hipGetErrorString(cuerr);
        payload.error_code = kCudaMemcpy;
        break;
      }
    }
  }

  // Wait for all compute and memcpy to complete.
  hipError_t cuerr = hipStreamSynchronize(stream_);
  if (cuerr != hipSuccess) {
    LOG_ERROR << "failed to synchronize GPU for addsub: "
              << hipGetErrorString(cuerr);
    return kCudaExecute;
  }

  return ErrorCodes::Success;
}
#endif  // TRTIS_ENABLE_GPU

int
Context::Execute(
    const uint32_t payload_cnt, CustomPayload* payloads,
    CustomGetNextInputFn_t input_fn, CustomGetOutputFn_t output_fn)
{
  if (gpu_device_ == CUSTOM_NO_GPU_DEVICE) {
    return ExecuteCPU(payload_cnt, payloads, input_fn, output_fn);
  } else {
#ifndef TRTIS_ENABLE_GPU
    return kGpuNotSupported;
#else
    return ExecuteGPU(payload_cnt, payloads, input_fn, output_fn);
#endif  // !TRTIS_ENABLE_GPU
  }
}

}  // namespace addsub

// Creates a new addsub context instance
int
CustomInstance::Create(
    CustomInstance** instance, const std::string& name,
    const ModelConfig& model_config, int gpu_device,
    const CustomInitializeData* data)
{
  addsub::Context* context =
      new addsub::Context(name, model_config, gpu_device);

  *instance = context;

  if (context == nullptr) {
    return ErrorCodes::CreationFailure;
  }

  return context->Init();
}

}}}  // namespace nvidia::inferenceserver::custom
