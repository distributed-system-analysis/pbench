#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

////////////////////////////////////////////////////////////////////////////////////////////////////

// Guard conditions around the entire file.
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 700

////////////////////////////////////////////////////////////////////////////////////////////////////

#include "cutlass_unit_tests.h"
#include "tools/util/half.h"
#include "tools/test/unit/gemm/gemm_testbed.h"
#include "cutlass/gemm/wmma_gemm_traits.h"
#include "cutlass/gemm/wmma_gemm_epilogue.h"

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename GemmTraits, typename EpilogueTraits, typename LoadAccumulatorIterator>
__global__ void test_epilogue_kernel(
  typename EpilogueTraits::Params params,
  cutlass::Coord<3> problem,
  typename EpilogueTraits::AccumulatorScalar *accum_ptr,
  int ldm) {

  // Shared memory allocation
  __shared__ typename EpilogueTraits::SharedStorage shared_storage;

  //
  // Load accumulators from memory - normally, a GEMM would compute these
  //

  // Traits class defines tiling
  GemmTraits traits;

  int warp_id = (threadIdx.x / 32);
  cutlass::Coord<3> warp_offset = traits(warp_id);

  // Accumulator fragment
  typename EpilogueTraits::AccumulatorFragment accumulator;

  // Construct an out-of-band LoadIterator for accumulators to initialize them

  LoadAccumulatorIterator load_accum_iterator(accum_ptr, ldm, warp_offset);
  load_accum_iterator.load(accumulator);

  __syncthreads();

  //
  // Test the epilogue itself
  //

  typedef cutlass::gemm::WmmaGemmEpilogue<EpilogueTraits> Epilogue;

  Epilogue epilogue(params, problem, warp_offset);

  // Perform the epilogue operation
  epilogue.update(shared_storage, accumulator);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <
  typename ThreadBlockTile,
  typename WarpTile,
  typename WmmaTile,
  typename EpilogueTile,
  typename StreamTile,
  typename AccumulatorType,
  typename ScalarC
>
struct TestWmmaGemmEpilogue {

  typedef cutlass::gemm::WmmaGemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    ThreadBlockTile,
    WarpTile,
    WmmaTile,
    AccumulatorType,
    AccumulatorType,
    1,
    AccumulatorType,
    EpilogueTile,
    StreamTile
  > Traits;

  // Construct an actual epilogue
  typedef cutlass::gemm::EpilogueLinearScaling<ScalarC, ScalarC, ScalarC, ScalarC> EpilogueLinearScaling;

  /// Define some traits
  typedef cutlass::gemm::WmmaGemmEpilogueTraitsBasic<
    ScalarC,
    typename Traits::WarpMultiplyAdd::StoreIteratorC,
    ScalarC,
    ThreadBlockTile,
    32 * Traits::Warps::kCount,
    WarpTile,
    WmmaTile,
    EpilogueTile,
    StreamTile,
    EpilogueLinearScaling
  > WmmaGemmEpilogueTraits;

  /// Type alias for EpilogueTraits type
  typedef typename WmmaGemmEpilogueTraits::Traits EpilogueTraits;

  TestWmmaGemmEpilogue() {

  }

  void run(cutlass::Coord<3> problem) {
    //
    // Prepare accumulator tile
    //
    cutlass::HostTensor<ScalarC> accumulator_matrix;
    cutlass::HostTensor<ScalarC> source_matrix;
    cutlass::HostTensor<ScalarC> destination_matrix;

    accumulator_matrix.resize_matrix(
      ThreadBlockTile::kW,
      ThreadBlockTile::kH,
      cutlass::MatrixLayout::kColumnMajor);

    source_matrix.resize_matrix(
      problem[2],
      problem[1],
      cutlass::MatrixLayout::kColumnMajor);

    destination_matrix.resize_matrix(
      problem[2],
      problem[1],
      cutlass::MatrixLayout::kColumnMajor);

    accumulator_matrix.fill_sequential();

    source_matrix.fill_sequential();

    int value = 0;
    for (int row = 0; row < ThreadBlockTile::kW; ++row) {
      for (int col = 0; col < ThreadBlockTile::kH; ++col, ++value) {
        if (row < problem[2] && col < problem[1]) {
          source_matrix.at(cutlass::make_Coord(0, row, col, 0)) = ScalarC(value);
        }
      }
    }

    destination_matrix.fill(0);

    //
    // Launch test kernel
    //
    dim3 grid(1,1);
    dim3 block(32 * Traits::Warps::kCount, 1, 1);

    EpilogueLinearScaling functor;
    functor.initialize(1, 0);

    typename EpilogueTraits::Params params;

    params.initialize(
      functor,
      source_matrix.device_data(),
      source_matrix.leading_dim(),
      destination_matrix.device_data(),
      destination_matrix.leading_dim()
    );

    test_epilogue_kernel<
      Traits,
      EpilogueTraits,
      typename Traits::WarpMultiplyAdd::LoadIteratorC
    ><<< grid, block >>>(
      params,
      problem,
      accumulator_matrix.device_data(),
      accumulator_matrix.leading_dim()
    );

    destination_matrix.sync_host();

    EXPECT_TRUE(accumulator_matrix.bit_equals(destination_matrix))
      << "Accumulators:\n" << accumulator_matrix << "\nDestination:\n" << destination_matrix;
  }

  void run() {
    run(cutlass::make_Coord(ThreadBlockTile::kD, ThreadBlockTile::kH, ThreadBlockTile::kW));
  }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

/// Small epilogue
TEST(WmmaGemm_16x16x16, wmma_epilogue_basic) {

  // GEMM threadblock structure
  typedef cutlass::Shape<16, 16, 16> ThreadBlockTile;
  typedef cutlass::Shape<16, 16, 16> WarpTile;
  typedef cutlass::Shape<16, 16, 16> WmmaTile;

  // Epilogue shapes
  typedef cutlass::Shape<1, 16, 16> EpilogueTile;
  typedef cutlass::Shape<1, 16, 16> StreamTile;

  typedef float AccumulatorType;
  typedef float ScalarC;

  TestWmmaGemmEpilogue<
    ThreadBlockTile,
    WarpTile,
    WmmaTile,
    EpilogueTile,
    StreamTile,
    AccumulatorType,
    ScalarC
  >().run();
}

TEST(WmmaGemm_16x16x16, wmma_epilogue_ragged) {

  // GEMM threadblock structure
  typedef cutlass::Shape<16, 16, 16> ThreadBlockTile;
  typedef cutlass::Shape<16, 16, 16> WarpTile;
  typedef cutlass::Shape<16, 16, 16> WmmaTile;

  // Epilogue shapes
  typedef cutlass::Shape<1, 16, 16> EpilogueTile;
  typedef cutlass::Shape<1, 16, 16> StreamTile;

  typedef float AccumulatorType;
  typedef float ScalarC;

  TestWmmaGemmEpilogue<
    ThreadBlockTile,
    WarpTile,
    WmmaTile,
    EpilogueTile,
    StreamTile,
    AccumulatorType,
    ScalarC
  >().run(cutlass::make_Coord(0, 15, 15));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

/// Small epilogue
TEST(WmmaGemm_32x32x16, wmma_epilogue_basic_32x32_32x32) {

  // GEMM threadblock structure
  typedef cutlass::Shape<16, 32, 32> ThreadBlockTile;
  typedef cutlass::Shape<16, 32, 32> WarpTile;
  typedef cutlass::Shape<16, 16, 16> WmmaTile;

  // Epilogue shapes
  typedef cutlass::Shape<1, 32, 32> EpilogueTile;
  typedef cutlass::Shape<1, 4, 32> StreamTile;

  typedef float AccumulatorType;
  typedef float ScalarC;

  TestWmmaGemmEpilogue<
    ThreadBlockTile,
    WarpTile,
    WmmaTile,
    EpilogueTile,
    StreamTile,
    AccumulatorType,
    ScalarC
  >().run();
}

/// Small epilogue
TEST(WmmaGemm_32x32x16, wmma_epilogue_basic_32x32_32x32_ragged) {

  // GEMM threadblock structure
  typedef cutlass::Shape<16, 32, 32> ThreadBlockTile;
  typedef cutlass::Shape<16, 32, 32> WarpTile;
  typedef cutlass::Shape<16, 16, 16> WmmaTile;

  // Epilogue shapes
  typedef cutlass::Shape<1, 32, 32> EpilogueTile;
  typedef cutlass::Shape<1, 4, 32> StreamTile;

  typedef float AccumulatorType;
  typedef float ScalarC;

  TestWmmaGemmEpilogue<
    ThreadBlockTile,
    WarpTile,
    WmmaTile,
    EpilogueTile,
    StreamTile,
    AccumulatorType,
    ScalarC
  >().run(cutlass::make_Coord(0, 14, 17));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

/// Small epilogue
TEST(WmmaGemm_32x32x16, wmma_epilogue_basic_32x32_16x16) {

  // GEMM threadblock structure
  typedef cutlass::Shape<16, 32, 32> ThreadBlockTile;
  typedef cutlass::Shape<16, 16, 16> WarpTile;
  typedef cutlass::Shape<16, 16, 16> WmmaTile;

  // Epilogue shapes
  typedef cutlass::Shape<1, 32, 32> EpilogueTile;
  typedef cutlass::Shape<1, 4, 32> StreamTile;

  typedef float AccumulatorType;
  typedef float ScalarC;

  TestWmmaGemmEpilogue<
    ThreadBlockTile,
    WarpTile,
    WmmaTile,
    EpilogueTile,
    StreamTile,
    AccumulatorType,
    ScalarC
  >().run();
}

/// Small epilogue
TEST(WmmaGemm_32x32x16, wmma_epilogue_basic_32x32_16x16_ragged) {

  // GEMM threadblock structure
  typedef cutlass::Shape<16, 32, 32> ThreadBlockTile;
  typedef cutlass::Shape<16, 16, 16> WarpTile;
  typedef cutlass::Shape<16, 16, 16> WmmaTile;

  // Epilogue shapes
  typedef cutlass::Shape<1, 32, 32> EpilogueTile;
  typedef cutlass::Shape<1, 4, 32> StreamTile;

  typedef float AccumulatorType;
  typedef float ScalarC;

  TestWmmaGemmEpilogue<
    ThreadBlockTile,
    WarpTile,
    WmmaTile,
    EpilogueTile,
    StreamTile,
    AccumulatorType,
    ScalarC
  >().run(cutlass::make_Coord(0, 23, 19));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

/// Large epilogue
TEST(WmmaGemm_128x128x16, wmma_epilogue_basic_32x32_16x16) {

  // GEMM threadblock structure
  typedef cutlass::Shape<16, 128, 128> ThreadBlockTile;
  typedef cutlass::Shape<16, 32, 64> WarpTile;
  typedef cutlass::Shape<16, 16, 16> WmmaTile;

  // Epilogue shapes
  typedef cutlass::Shape<1, 64, 64> EpilogueTile;
  typedef cutlass::Shape<1, 4, 64> StreamTile;

  typedef float AccumulatorType;
  typedef float ScalarC;

  typedef cutlass::gemm::WmmaGemmEpilogueStructure<
    ThreadBlockTile,
    EpilogueTile,
    StreamTile,
    WarpTile,
    WmmaTile
  > Structure;

  TestWmmaGemmEpilogue<
    ThreadBlockTile,
    WarpTile,
    WmmaTile,
    EpilogueTile,
    StreamTile,
    AccumulatorType,
    ScalarC
  >().run();
}

/// Large epilogue
TEST(WmmaGemm_128x128x16, wmma_epilogue_basic_32x32_16x16_ragged) {

  // GEMM threadblock structure
  typedef cutlass::Shape<16, 128, 128> ThreadBlockTile;
  typedef cutlass::Shape<16, 32, 64> WarpTile;
  typedef cutlass::Shape<16, 16, 16> WmmaTile;

  // Epilogue shapes
  typedef cutlass::Shape<1, 64, 64> EpilogueTile;
  typedef cutlass::Shape<1, 4, 64> StreamTile;

  typedef float AccumulatorType;
  typedef float ScalarC;

  typedef cutlass::gemm::WmmaGemmEpilogueStructure<
    ThreadBlockTile,
    EpilogueTile,
    StreamTile,
    WarpTile,
    WmmaTile
  > Structure;

  TestWmmaGemmEpilogue<
    ThreadBlockTile,
    WarpTile,
    WmmaTile,
    EpilogueTile,
    StreamTile,
    AccumulatorType,
    ScalarC
  >().run(cutlass::make_Coord(0, 119, 101));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#endif // end guard conditional on SM70
